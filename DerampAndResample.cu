#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#define __CUDA_INTERNAL_COMPILATION__
#include <math_functions.h>
#include <hip/hip_math_constants.h>
#include <hip/device_functions.h>
#include <hip/hip_complex.h>
//#include <hip/hip_runtime_api.h>
#include <cuda_occupancy.h>
#include "cuConstants.cuh"
#undef  __CUDA_INTERNAL_COMPILATION__
#include <math.h>
#include<time.h>
#include<complex>
#include <iostream>





#include <stdio.h>

using namespace std;

extern "C" void DerampDemodResample(
	complex<short>*SlaveArray,
	double *CpmAz,
	double *CpmRg,
	double AzimuthShift,
	complex<float>* output,
	float *KernelAz,
	float *KernelRg,
	int sBurstIdx,
	int slave_pixels,
	int slave_lines,
	int MasterBox[4],
	int SlaveBox[4],
	int S_linesPerBurst,
	int S_SamplesPerBurst,
	double azimuthTimeInterval,
	double* dopplerRate,
	double* referenceTime,
	double* dopplerCentroid,
	int Npoints
	);

extern "C" hipComplex* DerampDemodResample_ESD(
	complex<short>*SlaveArray,
	double *CpmAz,
	double *CpmRg,
	double AzimuthShift,
	complex<float>* output,
	float *KernelAz,
	float *KernelRg,
	int sBurstIdx,
	int slave_pixels,
	int slave_lines,
	int MasterBox[4],
	int SlaveBox[4],
	int linesPerBurst,
	int SamplesPerBurst,
	double azimuthTimeInterval,
	double* dopplerRate,
	double* referenceTime,
	double* dopplerCentroid,
	int Npoints
	);

extern "C" hipComplex* ResampleFirstBurst(
	complex<float>*SlaveArray,
	int ww,
	int hh
	);

__device__  hipComplex PartialCompute(hipComplex* Input, hipComplex *kernel)
{
	hipComplex tempAdd = make_hipComplex(0.0f, 0.0f);
	hipComplex tempMul;

	for (int i = 0; i < npoints; i++)
	{
		tempMul = hipCmulf(Input[i], kernel[i]);
		tempAdd = hipCaddf(tempAdd, tempMul);
	}

	return tempAdd;

}

__device__ inline double d_sqr(double x)
{
	return x*x;
}
__device__  inline hipComplex CmulfFloat(hipComplex Left, float Right)
{
	hipComplex Res;


	Res.x = Left.x*Right;
	Res.y = Left.y*Right;

	return Res;

}

__device__ inline void GetIndexes(double x, int Indexed[2])
{
	Indexed[0] = int(x);
	Indexed[1] = int((x - Indexed[0]) * c_Interval + 0.5);
}




__global__ void DerampDemod_Shared(float *PhaseArray, hipComplex* SlaveArray,
	double* dopplerRate, double *referenceTime, double* dopplerCentroid,
	int x0, int y0, int numLines, int numPixels, int firstLineInBurst,
	double azimuthTimeInterval, size_t d_pitch1,
	size_t d_pitch2, size_t d_pitch3, short2* SlaveArrayS2, size_t d_pitchS2)
{
	__shared__ double tile_Doppler[16];
	__shared__ double tile_ReferenceTime[16];
	__shared__ double tile_DopplerCentroid[16];


	const int row = blockIdx.y*blockDim.y + threadIdx.y;
	const int col = blockIdx.x*blockDim.x + threadIdx.x;

	int y = row + y0;
	int x = col + x0;


	if (threadIdx.y == 0)
	{

		tile_Doppler[threadIdx.x] = dopplerRate[x];
		tile_ReferenceTime[threadIdx.x] = referenceTime[x];
		tile_DopplerCentroid[threadIdx.x] = dopplerCentroid[x];

	}



	__syncthreads();

	if (row < numLines&&col < numPixels)
	{

		double ta = (y - firstLineInBurst)*azimuthTimeInterval;



		double kt = tile_Doppler[threadIdx.x];
		double deramp = -HIP_PI*kt*d_sqr(ta - tile_ReferenceTime[threadIdx.x]);
		double demod = -2.0 * HIP_PI*tile_DopplerCentroid[threadIdx.x] * ta;
		double phase = deramp + demod;




		float* rowPhaseArray = (float *)((char*)PhaseArray + row*d_pitch1);
		rowPhaseArray[col] = phase;


		hipComplex* rowSlaveArrayOutput = (hipComplex *)((char*)SlaveArray + row*d_pitch2);
		short2* rowSlaveArrayInput = (short2 *)((char*)SlaveArrayS2 + row*d_pitchS2);

		double SinCos[2];
		sincos(phase, SinCos, SinCos + 1);


		rowSlaveArrayOutput[col] = hipCmulf(make_hipComplex(rowSlaveArrayInput[col].x, rowSlaveArrayInput[col].y),
			make_hipComplex(SinCos[1], SinCos[0]));

		/*if (row == 50 && col == 2000)
		{
			printf("Deramped Slave:(%lf,%lf)\n", rowSlaveArrayOutput[col].x, rowSlaveArrayOutput[col].y);
		}*/
	}

}


__global__ void
resample_texture_kernel_6p(hipComplex *output, double* SlaveAz,
double* SlaveRg, size_t CorrPitch_1)
{
	__shared__ double SslaveRg[16][16];
	__shared__ double SslaveAz[16][16];

	const int row = blockIdx.y*blockDim.y + threadIdx.y;
	const int col = blockIdx.x*blockDim.x + threadIdx.x;


	//const int index = row*(overlap_pixehi - overlap_pixelo + 1) + col;
	hipComplex *rowoutput = (hipComplex *)((char*)output + row*CorrPitch_1);


	// adjust pCorr to point to row
	//output = (float *)((char*)output + row*CorrPitch);

	int Npoints2m1 = 6 / 2 - 1;



	if (row < c_mLines && col < c_mPixels)
	{


		double *rowSlaveRg = (double *)((char*)SlaveRg + row*CorrPitch_1);
		double *rowSlaveAz = (double *)((char*)SlaveAz + row*CorrPitch_1);


		SslaveRg[threadIdx.y][threadIdx.x] = rowSlaveRg[col];
		SslaveAz[threadIdx.y][threadIdx.x] = rowSlaveAz[col] + d_AzimuthShift;

		__syncthreads();


		const int  fl_interpL = int(SslaveAz[threadIdx.y][threadIdx.x]);
		const int fl_interpP = int(SslaveRg[threadIdx.y][threadIdx.x]);




		if (fl_interpL > c_sYmax || fl_interpL<c_sY0
			|| fl_interpP>c_sXmax || fl_interpP < c_sX0)
		{
			rowoutput[col] = make_hipComplex(0.0f, 0.0f);

		}

		else if (fl_interpL>c_sYmax - Npoints2m1 - 2 || fl_interpL<c_sY0 + Npoints2m1
			|| fl_interpP>c_sXmax - Npoints2m1 - 2 || fl_interpP < c_sX0 + Npoints2m1)
		{
			int indexL = fl_interpL - c_sY0;
			int indexP = fl_interpP - c_sX0;



			hipComplex SlaveArray = tex2D(tex_slave, indexP, indexL);
			float sampleI = hipCrealf(SlaveArray);
			float sampleQ = hipCimagf(SlaveArray);
			float samplePhase = tex2D(tex_PhaseArray, indexP, indexL);

			double ReSampleI = sampleI*cos(samplePhase) + sampleQ*sin(samplePhase);
			double ReSampleQ = -sampleI*sin(samplePhase) + sampleQ*cos(samplePhase);

			rowoutput[col] = make_hipComplex(ReSampleI, ReSampleQ);





		}
		else
		{
			const int indexL = fl_interpL - c_sY0;
			const int indexP = fl_interpP - c_sX0;

			const float interpLdec = SslaveAz[threadIdx.y][threadIdx.x] - fl_interpL; //shared[offset1];//fl_interpL;            // e.g. .35432
			const float interpPdec = SslaveRg[threadIdx.y][threadIdx.x] - fl_interpP;//shared[offset1 + 1];//fl_interpP; // e.g. .5232
			//float* rowPhaseArray = (float*)((char*)PhaseArray + indexL*Spitch);


			const int kernelnoL = int(interpLdec * 2047 + 0.5); // lookup table index
			const int kernelnoP = int(interpPdec * 2047 + 0.5); // lookup table index


			hipComplex kernelL[6], kernelP[6];
#pragma unroll 2
			for (int i = 0; i < 6; ++i)
			{
				kernelL[i].x = tex2D(tex_kernelAz, i, kernelnoL);
				kernelP[i].x = tex2D(tex_kernelRg, i, kernelnoP);
				//Axis[i] = tex2D(tex_Axis, i, kernelnoL);
			}

			//azimuth_shift(kernelL, Axis, interpP, d_rsr2x, d_f_DC_a0, d_f_DC_a1, d_f_DC_a2, d_slave_prf, npoints);





			hipComplex B[36];
			hipComplex Phase[36];
			float temp;
			//B[0] = tex2D(tex_slave, fl_interpP-slave_pixelo-3, fl_interpL-slave_linelo-3);
#pragma unroll 2
			for (int j = 0; j < 6; ++j)
			{
				for (int i = 0; i < 6; ++i)
				{
					B[j * 6 + i] = tex2D(tex_slave, indexP - Npoints2m1 + i, indexL - Npoints2m1 + j);
					temp = tex2D(tex_PhaseArray, indexP - Npoints2m1 + i, indexL - Npoints2m1 + j);
					Phase[j * 6 + i] = make_hipComplex(temp, 0.0f);

				}
			}


			hipComplex sum = make_hipComplex(0.0f, 0.0f);
			hipComplex sum1 = make_hipComplex(0.0f, 0.0f);
			hipComplex Result[6];
			hipComplex Result1[6];

#pragma unroll 2
			for (int jj = 0; jj < 6; ++jj)
			{
				for (int k = 0; k < 6; ++k)
				{
					sum = hipCaddf(sum, hipCmulf(kernelP[k], B[jj * 6 + k]));
					sum1 = hipCaddf(sum1, hipCmulf(kernelP[k], Phase[jj * 6 + k]));

				}
				Result[jj] = sum;
				Result1[jj] = sum1;
				sum = make_hipComplex(0.0f, 0.0f);// complex requires this
				sum1 = make_hipComplex(0.0f, 0.0f);
			}

			/*�о��������ս��*/
			sum = hipCaddf(hipCaddf(hipCaddf(hipCaddf(hipCaddf(hipCmulf(Result[0], kernelL[0]), hipCmulf(Result[1], kernelL[1])), hipCmulf(Result[2], kernelL[2])), hipCmulf(Result[3], kernelL[3])),
				hipCmulf(Result[4], kernelL[4])), hipCmulf(Result[5], kernelL[5]));
			sum1 = hipCaddf(hipCaddf(hipCaddf(hipCaddf(hipCaddf(hipCmulf(Result1[0], kernelL[0]), hipCmulf(Result1[1], kernelL[1])), hipCmulf(Result1[2], kernelL[2])), hipCmulf(Result1[3], kernelL[3])),
				hipCmulf(Result1[4], kernelL[4])), hipCmulf(Result1[5], kernelL[5]));


			float sampleI = hipCrealf(sum);
			float sampleQ = hipCimagf(sum);
			float samplePhase = hipCrealf(sum1);

			float ReSampleI = sampleI*cos(samplePhase) + sampleQ*sin(samplePhase);
			float ReSampleQ = -sampleI*sin(samplePhase) + sampleQ*cos(samplePhase);

			rowoutput[col] = make_hipComplex(ReSampleI, ReSampleQ);


		}
	}

}



__global__ void
resample_texture_kernel_12p_overlap_warpFunction
(hipComplex *output, int LineOffset, int Lines, size_t CorrPitch_1)
{

	const int row = blockIdx.y*blockDim.y + threadIdx.y;
	const int col = blockIdx.x*blockDim.x + threadIdx.x;

	hipComplex * rowoutput = (hipComplex *)((char*)output + row*CorrPitch_1);
	
	


	if (row < Lines && col < c_mPixels)
	{
		
		double Temp[2];
		Temp[0] = normalizeWarp((double)col, c_MasterBox[0], c_MasterBox[1]);
		Temp[1] = normalizeWarp((double)(row + LineOffset + c_mY0), c_MasterBox[2], c_MasterBox[3]);

		


	
		
		int IndexesL[2];//[interger index, decimal index]
		int IndexesP[2];
		

		GetIndexes(my_polyval(Temp[1], Temp[0], c_CpmAz) + d_AzimuthShift, IndexesL);
		GetIndexes(my_polyval(Temp[1], Temp[0], c_CpmRg), IndexesP);

		
		/*if (row == 100 && col == 1000)
		{
			
			printf("c_MasterBox[0]:%d\n", c_MasterBox[0]);
			printf("c_MasterBox[1]:%d\n", c_MasterBox[1]);
			printf("c_MasterBox[2]:%d\n", c_MasterBox[2]);
			printf("c_MasterBox[3]:%d\n", c_MasterBox[3]);
		}*/


		if (IndexesL[0] > c_sYmax || IndexesL[0]<c_sY0
			|| IndexesP[0]>c_sXmax || IndexesP[0] < c_sX0)
		{
			rowoutput[col] = make_hipComplex(0.0f, 0.0f);

		}

		else if (IndexesL[0]>c_sYmax - c_Npoints2m1 - 2 || IndexesL[0]<c_sY0 + c_Npoints2m1
			|| IndexesP[0]>c_sXmax - c_Npoints2m1 - 2 || IndexesP[0] < c_sX0 + c_Npoints2m1)
		{
			IndexesP[0] -= c_sX0;
			IndexesL[0] -= c_sY0;
			hipComplex SlaveArray = tex2D(tex_slave, IndexesP[0], IndexesL[0]);
			float samplePhase = tex2D(tex_PhaseArray, IndexesP[0] , IndexesL[0] );

			sincos(samplePhase, Temp, Temp + 1);
			rowoutput[col] = hipCmulf(SlaveArray, make_hipComplex(Temp[1], -Temp[0]));



		}
		else
		{
			
			
			//Read look-up tables of interpolation convolution kernels 
			float kernelL[12];
			float kernelP[12];

			
			kernelL[0] = tex2D(tex_kernelAz, 0, IndexesL[1]);
			kernelL[1] = tex2D(tex_kernelAz, 1, IndexesL[1]);
			kernelL[2] = tex2D(tex_kernelAz, 2, IndexesL[1]);
			kernelL[3] = tex2D(tex_kernelAz, 3, IndexesL[1]);
			kernelL[4] = tex2D(tex_kernelAz, 4, IndexesL[1]);
			kernelL[5] = tex2D(tex_kernelAz, 5, IndexesL[1]);
			kernelL[6] = tex2D(tex_kernelAz, 6, IndexesL[1]);
			kernelL[7] = tex2D(tex_kernelAz, 7, IndexesL[1]);
			kernelL[8] = tex2D(tex_kernelAz, 8, IndexesL[1]);
			kernelL[9] = tex2D(tex_kernelAz, 9, IndexesL[1]);
			kernelL[10] = tex2D(tex_kernelAz, 10, IndexesL[1]);
			kernelL[11] = tex2D(tex_kernelAz, 11, IndexesL[1]);

			kernelP[0] = tex2D(tex_kernelRg, 0, IndexesP[1]);
			kernelP[1] = tex2D(tex_kernelRg, 1, IndexesP[1]);
			kernelP[2] = tex2D(tex_kernelRg, 2, IndexesP[1]);
			kernelP[3] = tex2D(tex_kernelRg, 3, IndexesP[1]);
			kernelP[4] = tex2D(tex_kernelRg, 4, IndexesP[1]);
			kernelP[5] = tex2D(tex_kernelRg, 5, IndexesP[1]);
			kernelP[6] = tex2D(tex_kernelRg, 6, IndexesP[1]);
			kernelP[7] = tex2D(tex_kernelRg, 7, IndexesP[1]);
			kernelP[8] = tex2D(tex_kernelRg, 8, IndexesP[1]);
			kernelP[9] = tex2D(tex_kernelRg, 9, IndexesP[1]);
			kernelP[10] = tex2D(tex_kernelRg, 10, IndexesP[1]);
			kernelP[11] = tex2D(tex_kernelRg, 11, IndexesP[1]);


			hipComplex tempComplex;


			hipComplex sum = make_hipComplex(0.0f, 0.0f);
		
			double sum1 = 0.0;
			double tempFloat;

			IndexesP[0] -=(c_sX0+ c_Npoints2m1);
			IndexesL[0] -=(c_sY0+ c_Npoints2m1);

			
			
			//if (row == 100 && col == 1000)
			//{
			//	/*	printf("Resampled:(%lf,%lf)\n", rowoutput[col].x, rowoutput[col].y);
			//	printf("sum:(%lf,%lf)\n", sum.x, sum.y);
			//	printf("sum1:%lf\n", sum1);
			//	printf("sin:%lf\n", Temp[0]);
			//	printf("cos:%lf\n", Temp[1]);*/
			//	printf("IndexP:%d\n", IndexesP[0]);
			//	printf("IndexL:%d\n", IndexesL[0]);
			//	printf("Slave:(%lf,%lf)\n", tex2D(tex_slave, IndexesP[0], IndexesL[0]).x, 
			//		tex2D(tex_slave, IndexesP[0], IndexesL[0]).y);
			//	printf("Phase:%lf\n", tex2D(tex_PhaseArray, IndexesP[0], IndexesL[0]));
			//}



			// Partially unroll the loop to reduce register pressure
			// Interpolate the slave image at first
			for (int j = 0; j < 12; ++j)
			{
				tempComplex =
					hipCaddf(CmulfFloat(tex2D(tex_slave, IndexesP[0] + 11, IndexesL[0] + j), kernelP[11]),
					hipCaddf(CmulfFloat(tex2D(tex_slave, IndexesP[0] + 10, IndexesL[0] + j), kernelP[10]),
					hipCaddf(CmulfFloat(tex2D(tex_slave, IndexesP[0] + 9, IndexesL[0] + j), kernelP[9]),
					hipCaddf(CmulfFloat(tex2D(tex_slave, IndexesP[0] + 8, IndexesL[0] + j), kernelP[8]),
					hipCaddf(CmulfFloat(tex2D(tex_slave, IndexesP[0] + 7, IndexesL[0] + j), kernelP[7]),
					hipCaddf(CmulfFloat(tex2D(tex_slave, IndexesP[0] + 6, IndexesL[0] + j), kernelP[6]),
					hipCaddf(CmulfFloat(tex2D(tex_slave, IndexesP[0] + 5, IndexesL[0] + j), kernelP[5]),
					hipCaddf(CmulfFloat(tex2D(tex_slave, IndexesP[0] + 4, IndexesL[0] + j), kernelP[4]),
					hipCaddf(CmulfFloat(tex2D(tex_slave, IndexesP[0] + 3, IndexesL[0] + j), kernelP[3]),
					hipCaddf(CmulfFloat(tex2D(tex_slave, IndexesP[0] + 2, IndexesL[0] + j), kernelP[2]),
					hipCaddf(CmulfFloat(tex2D(tex_slave, IndexesP[0], IndexesL[0] + j), kernelP[0]),
					CmulfFloat(tex2D(tex_slave, IndexesP[0] + 1, IndexesL[0] + j), kernelP[1])
					)))))))))));

				sum = hipCaddf(sum, CmulfFloat(tempComplex, kernelL[j]));
			}



			//Interpolate the deramping phase
			for (int j = 0; j < 12; ++j)
			{

				tempFloat = (double)tex2D(tex_PhaseArray, IndexesP[0], IndexesL[0] + j)* kernelP[0]
					+ (double)tex2D(tex_PhaseArray, IndexesP[0] + 1, IndexesL[0] + j)*kernelP[1]
					+ (double)tex2D(tex_PhaseArray, IndexesP[0] + 2, IndexesL[0] + j)*kernelP[2]
					+ (double)tex2D(tex_PhaseArray, IndexesP[0] + 3, IndexesL[0] + j)*kernelP[3]
					+ (double)tex2D(tex_PhaseArray, IndexesP[0] + 4, IndexesL[0] + j)*kernelP[4]
					+ (double)tex2D(tex_PhaseArray, IndexesP[0] + 5, IndexesL[0] + j)*kernelP[5]
					+ (double)tex2D(tex_PhaseArray, IndexesP[0] + 6, IndexesL[0] + j)*kernelP[6]
					+ (double)tex2D(tex_PhaseArray, IndexesP[0] + 7, IndexesL[0] + j)*kernelP[7]
					+ (double)tex2D(tex_PhaseArray, IndexesP[0] + 8, IndexesL[0] + j)*kernelP[8]
					+ (double)tex2D(tex_PhaseArray, IndexesP[0] + 9, IndexesL[0] + j)*kernelP[9]
					+ (double)tex2D(tex_PhaseArray, IndexesP[0] + 10, IndexesL[0] + j)*kernelP[10]
					+ (double)tex2D(tex_PhaseArray, IndexesP[0] + 11, IndexesL[0] + j)*kernelP[11];

				
				
				sum1 = sum1 + tempFloat * kernelL[j];
			


			}

		
			//Compute Sin(sum1) and Cos(sum1) and store them into the Temp Array
			sincos(sum1, Temp, Temp+1);
			
			//Rereamp and output aligned signals
			rowoutput[col] = hipCmulf(sum, make_hipComplex(Temp[1], -Temp[0]));

		
			//if (row == 100 && col == 1000)
			//{


			//	printf("Resampled:(%lf,%lf)\n", rowoutput[col].x, rowoutput[col].y);
			//	printf("sum:(%lf,%lf)\n", sum.x, sum.y);
			//	printf("sum1:%lf\n", sum1);
			//	printf("sin:%lf\n", sin(sum1));//Temp[0]);
			//	printf("cos:%lf\n", cos(sum1));//Temp[1]);
			//	printf("IndexP:%d\n", IndexesP[1]);
			//	//printf("IndexL:%d\n", IndexesL[0]);
			//	//printf("FirstLinesum:(%lf,%lf)\n", sum.x,sum.y);
			//	//printf("Phase:%lf\n", tex2D(tex_PhaseArray, IndexesP[0], IndexesL[0]));
			//}
			


		}
	}

}




    void DerampDemodResample(
	complex<short>*SlaveArray,
	double *CpmAz,
	double *CpmRg,
	double AzimuthShift,
	complex<float>* output,
	float *KernelAz,
	float *KernelRg,
	int sBurstIdx,
	int slave_pixels,
	int slave_lines,
	int MasterBox[4],
	int SlaveBox[4],
	int linesPerBurst,
	int SamplesPerBurst,
	double azimuthTimeInterval,
	double* dopplerRate,
	double* referenceTime,
	double* dopplerCentroid,
	int Npoints
	)
{

	int sLines = SlaveBox[3] - SlaveBox[2] + 1;
	int sPixels = SlaveBox[1] - SlaveBox[0] + 1;
	int mLines = MasterBox[3] - MasterBox[2]+1;
	int mPixels = MasterBox[1] - MasterBox[0] + 1;

	

	int Npoints2m1 = Npoints / 2 - 1;


	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc_1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	int sfirstLineInBurst = sBurstIdx*linesPerBurst;

	hipHostRegister(SlaveArray, sLines*sPixels*sizeof(short2), hipHostRegisterDefault);
	hipHostRegister(dopplerCentroid, SamplesPerBurst*sizeof(double), hipHostRegisterDefault);
	hipHostRegister(referenceTime, SamplesPerBurst*sizeof(double), hipHostRegisterDefault);
	hipHostRegister(dopplerRate, SamplesPerBurst*sizeof(double), hipHostRegisterDefault);
	hipHostRegister(output, mLines*mPixels*sizeof(hipComplex), hipHostRegisterDefault);
	hipHostRegister(KernelAz, 2048 * 12 * sizeof(float), hipHostRegisterDefault);
	hipHostRegister(KernelRg, 2048 * 12 * sizeof(float), hipHostRegisterDefault);


	//checkCudaErrors(hipHostRegister(PhaseArray, sLines*sPixels*sizeof(float), hipHostRegisterDefault));
	size_t d_pitch1, d_pitch2, d_pitch3, d_pitchS2;


	// It is worth to use another array to save complex<short>
	float* d_PhaseArray;
	hipMallocPitch((void**)&d_PhaseArray, &d_pitch1, sPixels*sizeof(float), sLines);

	short2* d_SlaveArrayS2;
	hipMallocPitch((void**)&d_SlaveArrayS2, &d_pitchS2, sPixels*sizeof(short2), sLines);
	hipComplex* d_SlaveArray;
	hipMallocPitch((void**)&d_SlaveArray, &d_pitch2, sPixels*sizeof(hipComplex), sLines);


	double* d_dopplerRate, *d_referenceTime, *d_dopplerCentroid;
	hipMallocPitch((void**)&d_dopplerRate, &d_pitch3, SamplesPerBurst*sizeof(double), 1);
	hipMallocPitch((void**)&d_referenceTime, &d_pitch3, SamplesPerBurst*sizeof(double), 1);
	hipMallocPitch((void**)&d_dopplerCentroid, &d_pitch3, SamplesPerBurst*sizeof(double), 1);


	size_t CorrPitch;

	hipComplex * d_resample;
	hipMallocPitch((void **)&d_resample, &CorrPitch, mPixels*sizeof(hipComplex), mLines);


	hipArray *KernelAzArray = NULL;
	hipArray *KernelRgArray = NULL;
	hipMallocArray(&KernelAzArray, &channelDesc_1, Npoints, 2048);
	hipMallocArray(&KernelRgArray, &channelDesc_1, Npoints, 2048);


	dim3 threads(16, 16);
	dim3 blocks = dim3((sPixels + 15) / 16, (sLines + 15) / 16);
	hipStream_t stream[4];
	for (int i = 0; i < 4; i++)
	{
		hipStreamCreate(&stream[i]);
	}
	hipEvent_t g_start, g_stop;
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(resample_texture_kernel_12p_overlap_warpFunction_test), hipFuncCachePreferL1);

	//Memcpy to Constant  Variables
	hipMemcpyToSymbol(HIP_SYMBOL(npoints), &Npoints, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_mLines), &mLines, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_mPixels), &mPixels, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_sX0), &SlaveBox[0], sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_sXmax), &SlaveBox[1], sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_sY0), &SlaveBox[2], sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_sYmax), &SlaveBox[3], sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_mY0), &MasterBox[2], sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_AzimuthShift), &AzimuthShift, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_MasterBox), MasterBox, 4 * sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_Npoints2m1), &Npoints2m1, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_CpmAz), CpmAz, 6 * sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_CpmRg), CpmRg, 6 * sizeof(double), 0, hipMemcpyHostToDevice);




	//checkCudaErrors(hipMemcpy2DAsync(d_SlaveArray, d_pitch2, SlaveArray, sPixels*sizeof(hipComplex), sPixels*sizeof(hipComplex), sLines, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpy2DAsync(d_dopplerRate, d_pitch3, dopplerRate, SamplesPerBurst*sizeof(double), SamplesPerBurst*sizeof(double), 1, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpy2DAsync(d_referenceTime, d_pitch3, referenceTime, SamplesPerBurst*sizeof(double), SamplesPerBurst*sizeof(double), 1, hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpy2DAsync(d_dopplerCentroid, d_pitch3, dopplerCentroid, SamplesPerBurst*sizeof(double), SamplesPerBurst*sizeof(double), 1, hipMemcpyHostToDevice));

	//checkCudaErrors(hipMemcpy2D(d_SlaveArray, d_pitch2, SlaveArray, sPixels*sizeof(hipComplex), sPixels*sizeof(hipComplex), sLines, hipMemcpyHostToDevice));
	hipMemcpy2D(d_SlaveArrayS2, d_pitchS2, SlaveArray, sPixels*sizeof(short2), sPixels*sizeof(short2), sLines, hipMemcpyHostToDevice);
	hipMemcpy2D(d_dopplerRate, d_pitch3, dopplerRate, SamplesPerBurst*sizeof(double), SamplesPerBurst*sizeof(double), 1, hipMemcpyHostToDevice);
	hipMemcpy2D(d_referenceTime, d_pitch3, referenceTime, SamplesPerBurst*sizeof(double), SamplesPerBurst*sizeof(double), 1, hipMemcpyHostToDevice);
	hipMemcpy2D(d_dopplerCentroid, d_pitch3, dopplerCentroid, SamplesPerBurst*sizeof(double), SamplesPerBurst*sizeof(double), 1, hipMemcpyHostToDevice);


	//DerampDemod_Shared << <blocks, threads >> >(d_PhaseArray, d_SlaveArray, d_dopplerRate, d_referenceTime, d_dopplerCentroid, sX0, sY0, sLines, sPixels, sfirstLineInBurst,
	//azimuthTimeInterval, d_pitch1, d_pitch2, d_pitch3);

	DerampDemod_Shared << <blocks, threads >> >(d_PhaseArray, d_SlaveArray, d_dopplerRate, d_referenceTime, d_dopplerCentroid, SlaveBox[0], SlaveBox[2], sLines, sPixels, sfirstLineInBurst,
		azimuthTimeInterval, d_pitch1, d_pitch2, d_pitch3, d_SlaveArrayS2, d_pitchS2);

	//hipEventRecord(g_stop, 0);
	//hipEventSynchronize(g_stop);
	//hipEventElapsedTime(&time_cost1, g_start, g_stop);
	//cout << "DeRamping duration:" << time_cost1 << endl;







	hipMemcpyToArray(KernelAzArray, 0, 0, KernelAz, Npoints * 2048 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToArray(KernelRgArray, 0, 0, KernelRg, Npoints * 2048 * sizeof(float), hipMemcpyHostToDevice);
	hipBindTextureToArray(tex_kernelAz, KernelAzArray, channelDesc_1);
	hipBindTextureToArray(tex_kernelRg, KernelRgArray, channelDesc_1);
	hipBindTexture2D(0, tex_PhaseArray, d_PhaseArray, channelDesc_1, sPixels, sLines, d_pitch1);
	hipBindTexture2D(0, tex_slave, d_SlaveArray, channelDesc, sPixels, sLines, d_pitch2);


	size_t SPitch, MPitch;


	threads = dim3(16, 16);
	blocks = dim3((mPixels + threads.x - 1) / threads.x,
		(mLines + threads.y - 1) / threads.y);



	//for Subsets
	int partMlines = mLines / 4;
	int RemainMlines = mLines % 4;
	dim3 Partblocks = dim3((mPixels + threads.x - 1) / threads.x,
		(partMlines + threads.y - 1) / threads.y);
	dim3 Lastblocks = dim3((mPixels + threads.x - 1) / threads.x,
		(partMlines + RemainMlines + threads.y - 1) / threads.y);

	int PartOffsetD = partMlines*CorrPitch / 8;
	int PartOffsetH = partMlines*mPixels;




	hipFuncSetCacheConfig(reinterpret_cast<const void*>(resample_texture_kernel_12p_overlap_warpFunction), hipFuncCachePreferL1);


	float time_cost1, time_cost2;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);


	if (Npoints == 6)
	{

	}


	if (Npoints == 12)
	{
		
		//SubSet1

		resample_texture_kernel_12p_overlap_warpFunction
		<< <Partblocks, threads, 0, stream[0] >> >(d_resample, 0,
			partMlines, CorrPitch);

		hipMemcpy2DAsync(output, (mPixels)*sizeof(hipComplex), d_resample, CorrPitch, (mPixels)*sizeof(hipComplex), partMlines, hipMemcpyDeviceToHost, stream[0]);



		//SubSet2
		resample_texture_kernel_12p_overlap_warpFunction
			<< <Partblocks, threads, 0, stream[1] >>>
			(d_resample + PartOffsetD, partMlines,
			partMlines, CorrPitch);
		hipMemcpy2DAsync(output + PartOffsetH, (mPixels)*sizeof(hipComplex), 
			d_resample + PartOffsetD, CorrPitch, (mPixels)*sizeof(hipComplex),
			partMlines, hipMemcpyDeviceToHost, stream[1]);

		//SubSet3
		
		resample_texture_kernel_12p_overlap_warpFunction
		<< <Partblocks, threads, 0, stream[2] >>>
		(d_resample + 2 * PartOffsetD, 2 * partMlines,
			partMlines, CorrPitch);
		hipMemcpy2DAsync(output + 2 * PartOffsetH, (mPixels)*sizeof(hipComplex),
			d_resample + 2 * PartOffsetD, CorrPitch, (mPixels)*sizeof(hipComplex),
			partMlines, hipMemcpyDeviceToHost, stream[2]);


		//SubSet4
		resample_texture_kernel_12p_overlap_warpFunction
			<< <Lastblocks, threads, 0, stream[3] >> >
			(d_resample + 3 * PartOffsetD, 3 * partMlines,
			partMlines + RemainMlines, CorrPitch);
		hipMemcpy2DAsync(output + 3 * PartOffsetH, (mPixels)*sizeof(hipComplex), 
			d_resample + 3 * PartOffsetD, CorrPitch, (mPixels)*sizeof(hipComplex), 
			(partMlines + RemainMlines), hipMemcpyDeviceToHost, stream[3]);

		

		

	}
	

	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	hipEventElapsedTime(&time_cost2, g_start, g_stop);
	cout << "kernel duration:" << time_cost2 << endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);

	


	for (int i = 0; i < 4; i++)
	{
		hipStreamDestroy(stream[i]);
	}



	hipHostUnregister(SlaveArray);
	hipHostUnregister(output);
	hipHostUnregister(dopplerCentroid);
	hipHostUnregister(dopplerRate);
	hipHostUnregister(referenceTime);
	hipHostUnregister(KernelAz);
	hipHostUnregister(KernelRg);

	hipUnbindTexture(tex_kernelAz);
	hipUnbindTexture(tex_kernelRg);
	hipUnbindTexture(tex_slave);
	hipUnbindTexture(tex_PhaseArray);

	hipFree(d_PhaseArray);
	hipFree(d_SlaveArray);
	hipFree(d_dopplerRate);
	hipFree(d_referenceTime);
	hipFree(d_dopplerCentroid);
	hipFree(d_resample);
	hipFreeArray(KernelAzArray);
	hipFreeArray(KernelRgArray);
	hipFree(d_SlaveArrayS2);



	hipDeviceReset();




}



hipComplex* DerampDemodResample_ESD(
	complex<short>*SlaveArray,
	double *CpmAz,
	double *CpmRg,
	double AzimuthShift,
	complex<float>* output,
	float *KernelAz,
	float *KernelRg,
	int sBurstIdx,
	int slave_pixels,
	int slave_lines,
	int MasterBox[4],
	int SlaveBox[4],
	int linesPerBurst,
	int SamplesPerBurst,
	double azimuthTimeInterval,
	double* dopplerRate,
	double* referenceTime,
	double* dopplerCentroid,
	int Npoints
	)
{


	int sLines = SlaveBox[3] - SlaveBox[2] + 1;
	int sPixels = SlaveBox[1] - SlaveBox[0] + 1;
	int mLines = MasterBox[3] - MasterBox[2] + 1;
	int mPixels = MasterBox[1] - MasterBox[0] + 1;




	int Npoints2m1 = Npoints / 2 - 1;


	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc_1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	int sfirstLineInBurst = sBurstIdx*linesPerBurst;

	hipHostRegister(SlaveArray, sLines*sPixels*sizeof(short2), hipHostRegisterDefault);
	hipHostRegister(dopplerCentroid, SamplesPerBurst*sizeof(double), hipHostRegisterDefault);
	hipHostRegister(referenceTime, SamplesPerBurst*sizeof(double), hipHostRegisterDefault);
	hipHostRegister(dopplerRate, SamplesPerBurst*sizeof(double), hipHostRegisterDefault);
	hipHostRegister(KernelAz, 2048 * 12 * sizeof(float), hipHostRegisterDefault);
	hipHostRegister(KernelRg, 2048 * 12 * sizeof(float), hipHostRegisterDefault);
	hipHostRegister(output, mLines*mPixels*sizeof(hipComplex), hipHostRegisterDefault);


	//checkCudaErrors(hipHostRegister(PhaseArray, sLines*sPixels*sizeof(float), hipHostRegisterDefault));
	size_t d_pitch1, d_pitch2, d_pitch3, d_pitchS2;


	// It is worth to use another array to save complex<short>
	float* d_PhaseArray;
	hipMallocPitch((void**)&d_PhaseArray, &d_pitch1, sPixels*sizeof(float), sLines);

	short2* d_SlaveArrayS2;
	hipMallocPitch((void**)&d_SlaveArrayS2, &d_pitchS2, sPixels*sizeof(short2), sLines);
	hipComplex* d_SlaveArray;
	hipMallocPitch((void**)&d_SlaveArray, &d_pitch2, sPixels*sizeof(hipComplex), sLines);


	double* d_dopplerRate, *d_referenceTime, *d_dopplerCentroid;
	hipMallocPitch((void**)&d_dopplerRate, &d_pitch3, SamplesPerBurst*sizeof(double), 1);
	hipMallocPitch((void**)&d_referenceTime, &d_pitch3, SamplesPerBurst*sizeof(double), 1);
	hipMallocPitch((void**)&d_dopplerCentroid, &d_pitch3, SamplesPerBurst*sizeof(double), 1);


	size_t CorrPitch;

	hipComplex * d_resample;
	hipMallocPitch((void **)&d_resample, &CorrPitch, mPixels*sizeof(hipComplex), mLines);


	hipArray *KernelAzArray = NULL;
	hipArray *KernelRgArray = NULL;
	hipMallocArray(&KernelAzArray, &channelDesc_1, Npoints, 2048);
	hipMallocArray(&KernelRgArray, &channelDesc_1, Npoints, 2048);


	dim3 threads(16, 16);
	dim3 blocks = dim3((sPixels + 15) / 16, (sLines + 15) / 16);
	hipStream_t stream[4];
	for (int i = 0; i < 4; i++)
	{
		hipStreamCreate(&stream[i]);
	}
	hipEvent_t g_start, g_stop;
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(resample_texture_kernel_12p_overlap_warpFunction_test), hipFuncCachePreferL1);

	//Memcpy to Constant  Variables
	hipMemcpyToSymbol(HIP_SYMBOL(npoints), &Npoints, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_mLines), &mLines, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_mPixels), &mPixels, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_sX0), &SlaveBox[0], sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_sXmax), &SlaveBox[1], sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_sY0), &SlaveBox[2], sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_sYmax), &SlaveBox[3], sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_mY0), &MasterBox[2], sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_AzimuthShift), &AzimuthShift, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_MasterBox), MasterBox, 4 * sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_Npoints2m1), &Npoints2m1, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_CpmAz), CpmAz, 6 * sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_CpmRg), CpmRg, 6 * sizeof(double), 0, hipMemcpyHostToDevice);




	

	
	hipMemcpy2D(d_SlaveArrayS2, d_pitchS2, SlaveArray, sPixels*sizeof(short2), sPixels*sizeof(short2), sLines, hipMemcpyHostToDevice);
	hipMemcpy2D(d_dopplerRate, d_pitch3, dopplerRate, SamplesPerBurst*sizeof(double), SamplesPerBurst*sizeof(double), 1, hipMemcpyHostToDevice);
	hipMemcpy2D(d_referenceTime, d_pitch3, referenceTime, SamplesPerBurst*sizeof(double), SamplesPerBurst*sizeof(double), 1, hipMemcpyHostToDevice);
	hipMemcpy2D(d_dopplerCentroid, d_pitch3, dopplerCentroid, SamplesPerBurst*sizeof(double), SamplesPerBurst*sizeof(double), 1, hipMemcpyHostToDevice);


	//DerampDemod_Shared << <blocks, threads >> >(d_PhaseArray, d_SlaveArray, d_dopplerRate, d_referenceTime, d_dopplerCentroid, sX0, sY0, sLines, sPixels, sfirstLineInBurst,
	//azimuthTimeInterval, d_pitch1, d_pitch2, d_pitch3);

	DerampDemod_Shared << <blocks, threads >> >(d_PhaseArray, d_SlaveArray, d_dopplerRate, d_referenceTime, d_dopplerCentroid, SlaveBox[0], SlaveBox[2], sLines, sPixels, sfirstLineInBurst,
		azimuthTimeInterval, d_pitch1, d_pitch2, d_pitch3, d_SlaveArrayS2, d_pitchS2);


	//hipEventRecord(g_stop, 0);
	//hipEventSynchronize(g_stop);
	//hipEventElapsedTime(&time_cost1, g_start, g_stop);
	//cout << "DeRamping duration:" << time_cost1 << endl;







	hipMemcpyToArray(KernelAzArray, 0, 0, KernelAz, Npoints * 2048 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToArray(KernelRgArray, 0, 0, KernelRg, Npoints * 2048 * sizeof(float), hipMemcpyHostToDevice);
	hipBindTextureToArray(tex_kernelAz, KernelAzArray, channelDesc_1);
	hipBindTextureToArray(tex_kernelRg, KernelRgArray, channelDesc_1);
	hipBindTexture2D(0, tex_PhaseArray, d_PhaseArray, channelDesc_1, sPixels, sLines, d_pitch1);
	hipBindTexture2D(0, tex_slave, d_SlaveArray, channelDesc, sPixels, sLines, d_pitch2);

	//float *d_KernelAz, *d_KernelRg;
	//hipMalloc((void**)&d_KernelAz, 2048 * Npoints*sizeof(float));
	//hipMalloc((void**)&d_KernelRg, 2048 * Npoints*sizeof(float));
	//checkCudaErrors(hipMemcpy(d_KernelAz, KernelAz, 2048 * Npoints*sizeof(float), hipMemcpyHostToDevice));
	//checkCudaErrors(hipMemcpy(d_KernelRg, KernelRg, 2048 * Npoints*sizeof(float), hipMemcpyHostToDevice));


	size_t SPitch, MPitch;


	threads = dim3(16, 16);
	blocks = dim3((mPixels + threads.x - 1) / threads.x,
		(mLines + threads.y - 1) / threads.y);



	//for Subsets
	int partMlines = mLines / 4;
	int RemainMlines = mLines % 4;
	dim3 Partblocks = dim3((mPixels + threads.x - 1) / threads.x,
		(partMlines + threads.y - 1) / threads.y);
	dim3 Lastblocks = dim3((mPixels + threads.x - 1) / threads.x,
		(partMlines + RemainMlines + threads.y - 1) / threads.y);

	int PartOffsetD = partMlines*CorrPitch / 8;
	int PartOffsetH = partMlines*mPixels;




	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(resample_texture_kernel_12p_overlap_warpFunction), hipFuncCachePreferL1);


	float time_cost1, time_cost2;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);


	if (Npoints == 6)
	{

	}


	if (Npoints == 12)
	{

		//SubSet1

		resample_texture_kernel_12p_overlap_warpFunction
			<< <Partblocks, threads, 0, stream[0] >> >(d_resample, 0,
			partMlines, CorrPitch);

		hipMemcpy2DAsync(output, (mPixels)*sizeof(hipComplex), d_resample, CorrPitch, (mPixels)*sizeof(hipComplex), partMlines, hipMemcpyDeviceToHost, stream[0]);



		//SubSet2
		resample_texture_kernel_12p_overlap_warpFunction
			<< <Partblocks, threads, 0, stream[1] >> >
			(d_resample + PartOffsetD, partMlines,
			partMlines, CorrPitch);
		hipMemcpy2DAsync(output + PartOffsetH, (mPixels)*sizeof(hipComplex),
			d_resample + PartOffsetD, CorrPitch, (mPixels)*sizeof(hipComplex),
			partMlines, hipMemcpyDeviceToHost, stream[1]);

		//SubSet3

		resample_texture_kernel_12p_overlap_warpFunction
			<< <Partblocks, threads, 0, stream[2] >> >
			(d_resample + 2 * PartOffsetD, 2 * partMlines,
			partMlines, CorrPitch);
		hipMemcpy2DAsync(output + 2 * PartOffsetH, (mPixels)*sizeof(hipComplex),
			d_resample + 2 * PartOffsetD, CorrPitch, (mPixels)*sizeof(hipComplex),
			partMlines, hipMemcpyDeviceToHost, stream[2]);


		//SubSet4
		resample_texture_kernel_12p_overlap_warpFunction
			<< <Lastblocks, threads, 0, stream[3] >> >
			(d_resample + 3 * PartOffsetD, 3 * partMlines,
			partMlines + RemainMlines, CorrPitch);
		hipMemcpy2DAsync(output + 3 * PartOffsetH, (mPixels)*sizeof(hipComplex),
			d_resample + 3 * PartOffsetD, CorrPitch, (mPixels)*sizeof(hipComplex),
			(partMlines + RemainMlines), hipMemcpyDeviceToHost, stream[3]);





	}
	//checkCudaErrors(hipMemcpy2D(output, (mPixels)*sizeof(hipComplex), d_resample, CorrPitch, (mPixels)*sizeof(hipComplex), mLines, hipMemcpyDeviceToHost));

	
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	hipEventElapsedTime(&time_cost2, g_start, g_stop);
	cout << "kernel duration:" << time_cost2 << endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);




	for (int i = 0; i < 4; i++)
	{
		hipStreamDestroy(stream[i]);
	}



	hipHostUnregister(SlaveArray);
	hipHostUnregister(output);
	hipHostUnregister(dopplerCentroid);
	hipHostUnregister(dopplerRate);
	hipHostUnregister(referenceTime);
	hipHostUnregister(KernelAz);
	hipHostUnregister(KernelRg);

	hipUnbindTexture(tex_kernelAz);
	hipUnbindTexture(tex_kernelRg);
	hipUnbindTexture(tex_slave);
	hipUnbindTexture(tex_PhaseArray);

	hipFree(d_PhaseArray);
	hipFree(d_SlaveArray);
	hipFree(d_dopplerRate);
	hipFree(d_referenceTime);
	hipFree(d_dopplerCentroid);
	//checkCudaErrors(hipFree(d_resample));
	hipFreeArray(KernelAzArray);
	hipFreeArray(KernelRgArray);
	hipFree(d_SlaveArrayS2);



	hipComplex *d_output = d_resample;
	return d_output;

}

hipComplex* ResampleFirstBurst(
	complex<float>*SlaveArray,
	int ww,
	int hh
	)
{
	
	//Page-Locking host Memory
	hipHostRegister(SlaveArray, ww*hh*sizeof(complex<float>), hipHostRegisterDefault);
	
	size_t RePitch;
	hipComplex * d_Reslave;
	hipMallocPitch((void **)&d_Reslave, &RePitch, ww*sizeof(hipComplex), hh);
	hipMemcpy2D(d_Reslave, RePitch, SlaveArray, 
		ww*sizeof(hipComplex), ww*sizeof(hipComplex), hh, hipMemcpyHostToDevice);

	hipHostUnregister(SlaveArray);
	return d_Reslave;
}